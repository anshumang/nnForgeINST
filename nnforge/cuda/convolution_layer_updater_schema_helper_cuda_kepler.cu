/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "convolution_layer_updater_schema_helper_cuda_kepler.h"

#include "convolution_layer_updater_cuda_kepler.cuh"

#include <boost/format.hpp>
#include "../neural_network_exception.h"

namespace nnforge
{
	namespace cuda
	{
		layer_updater_cuda_smart_ptr convolution_layer_updater_schema_helper_cuda_kepler::create_updater_specific(
				const layer_configuration_specific& input_configuration_specific,
				const layer_configuration_specific& output_configuration_specific)
		{
			layer_updater_cuda_smart_ptr res;

			switch (output_configuration_specific.dimension_sizes.size()) 
			{
				case 2:
					res = layer_updater_cuda_smart_ptr(new convolution_layer_updater_cuda_kepler<2>());
					break;
				case 3:
					res = layer_updater_cuda_smart_ptr(new convolution_layer_updater_cuda_kepler<3>());
					break;
				case 4:
					res = layer_updater_cuda_smart_ptr(new convolution_layer_updater_cuda_kepler<4>());
					break;
				default:
					throw neural_network_exception((boost::format("No CUDA updater for the convolutional layer of %1% dimensions for Kepler and above architectures") % output_configuration_specific.dimension_sizes.size()).str());
			}

			return res;
		}
	}
}
